#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <cupti_events.h>
#include <unistd.h>


int main(int argc, char *argv[])
{
  hipCtx_t context;
  hipDevice_t device;

  CUptiResult cuptiErr;
  CUpti_EventGroup eventGroup;
  CUpti_EventID eventId;

  const char *eventName;

  size_t bytesRead, valueSize;
  uint32_t numInstances = 0, j = 0;
  uint64_t *eventValues = NULL, eventVal = 0;
  uint32_t profile_all = 1;


  if (argc > 1)
    deviceNum = atoi(argv[1]);
  else
    deviceNum = 0;

  if (argc > 2) {
    eventName = argv[2];
  }
  else {
    eventName = "inst_executed";
  }

  
  hipDeviceGet(&device, deviceNum);

  hipCtxCreate(&context, 0, device);


  cuptiSetEventCollectionMode(context, CUPTI_EVENT_COLLECTION_MODE_CONTINUOUS);


  cuptiEventGroupCreate(context, &eventGroup, 0);


  cuptiEventGetIdFromName(device, eventName, &eventId);
  

  cuptiEventGroupAddEvent(eventGroup, eventId);
  

  cuptiEventGroupSetAttribute(eventGroup, CUPTI_EVENT_GROUP_ATTR_PROFILE_ALL_DOMAIN_INSTANCES, sizeof(profile_all), &profile_all);
  
  cuptiErr = cuptiEventGroupEnable(eventGroup);
 
  valueSize = sizeof(numInstances);
  cuptiErr = cuptiEventGroupGetAttribute(eventGroup, CUPTI_EVENT_GROUP_ATTR_INSTANCE_COUNT, &valueSize, &numInstances);

  bytesRead = sizeof(uint64_t) * numInstances;
  eventValues = (uint64_t *) malloc(bytesRead);

  if (eventValues == NULL) {
      printf("%s:%d: Failed to allocate memory.\n", __FILE__, __LINE__);
      exit(-1);
  }

  int i = 0;

  do {
    cuptiErr = cuptiEventGroupReadEvent(eventGroup,
                                        CUPTI_EVENT_READ_FLAG_NONE,
                                        eventId, &bytesRead, eventValues);
  
    if (bytesRead != (sizeof(uint64_t) * numInstances)) {
      printf("Failed to read value for \"%s\"\n", eventName);
      exit(-1);
    }

    for (j = 0; j < numInstances; j++) {
      eventVal += eventValues[j];
    }
    printf("%s: %llu\n", eventName, (unsigned long long)eventVal);
    sleep(1);
    i+=1;
  } while (i<=30);

  cuptiErr = cuptiEventGroupDisable(eventGroup);

  cuptiErr = cuptiEventGroupDestroy(eventGroup);

  free(eventValues);
  return 0;
}
